
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int *a, int *b, int *c){
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  //if (index < n)
  c[index] = a[index] + b[index];
}

void random_ints(int *p, int s){
  for(int i=0; i < s; i++){
    p[i] = rand();
  }
}

#define N (2048*2048)
#define THREADS_PER_BLOCK 512
int main(){
  int *a, *b, *c;  // host copies
  int *d_a, *d_b, *d_c;  //device copies
  int size = N * sizeof(int);
  
  // Space allocation for device copies
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);
  
  // Alloc space for host copies of a, b, c and setup input values
  a = (int *)malloc(size); random_ints(a, N);
  b = (int *)malloc(size); random_ints(b, N);
  c = (int *)malloc(size);
  
  // Copy inputs to Device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  
  // Launch add Kernel on GPU with N threads
  add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a, d_b, d_c);
  //add<<<(N + M-1) / M,M>>>(d_a, d_b, d_c, N);
  
  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  
  // Cleanup
  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  
  std::cout<<"Done!"<<std::endl;
  
  return 0;
}